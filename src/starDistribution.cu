#include "hip/hip_runtime.h"
/* starDistribution.cu
 * 
 * this module deals with snapshots of the galaxy: generation, retrieval of 
 * data from device and reduction into radial histograms.
 */


// ========================================================================= //
// dependencies

#include <ctime>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hiprand.h>

#include "globals.h"
#include "starDistribution.h"


// ========================================================================= //
// make galaxy

// ------------------------------------------------------------------------- //
// proc device

__global__ void makeGalaxyOnDevice_component() {
  /* assumes that all components of GALAXY have been initialized in the range
   * 0.0f..1.0f and rescales them according to the global constants
   */
  
  
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  
  if (i < N_STARS) {
    GALAXY[i].position.x -= 0.5;
    GALAXY[i].position.y -= 0.5;
    GALAXY[i].position.z -= 0.5;
    
    GALAXY[i].position.x *= D_UNIVERSE;
    GALAXY[i].position.y *= D_UNIVERSE;
    GALAXY[i].position.z *= D_UNIVERSE;
    
    GALAXY[i].velocity.x -= 0.5;
    GALAXY[i].velocity.y -= 0.5;
    GALAXY[i].velocity.z -= 0.5;
    
    GALAXY[i].velocity.x *= V_INIT_MAX;
    GALAXY[i].velocity.z *= V_INIT_MAX;
    GALAXY[i].velocity.z *= V_INIT_MAX;
    
    GALAXY[i].mass *= M_STAR_MAX;
  }
}

// ------------------------------------------------------------------------- //
// proc host

void makeGalaxyOnDevice() {
  /* This will initialize a galaxy with uniform distribution of
   * - position
   * - velocity
   * - mass
   * of the stars. This proc will place values 0.0f..1.0f into the given fields
   * and makes use of the below kernel to rescale/translate them into a 
   * reasonable range.
   * 
   * useful link:
   *   https://docs.nvidia.com/cuda/hiprand/host-api-overview.html
   */
  
  if (!flag_rand_initialized) {ABORT_WITH_MSG("RNG not initialized.");}
  
  // run the RNG
  hiprandGenerateUniform(
    d_RNG_mem, 
    (float *) d_galaxy,                             // galaxy is of type star *. star is a struct of only floats. This cast is justifiable.
    N_stars * (sizeof(star_t) / sizeof(float))
  );
  CudaCheckError();
  
  makeGalaxyOnDevice_component<<<nBlocks, blockSize>>>();
  hipDeviceSynchronize();
}

// ========================================================================= //
// get back galaxy from device

void fetchGalaxyFromDevice() {
  /* TODO: make this asynchronous
   * for this, you'll need to transform h_galaxy to a page locked array
   * use hipHostMalloc, cf. script p.33f.
   * 
   * This assumes that h_galaxy and d_galaxy have been properly initialized.
   * The "CTor" init() in globals.cu takes care of this.
   */
  
  hipMemcpy(
    h_galaxy, 
    d_galaxy, 
    N_stars * sizeof(star_t),
    hipMemcpyDeviceToHost
  );
  CudaCheckError();
}

// ========================================================================= //
// get all distances from a given star at index k

__global__ void makeDistanceComponent(unsigned int k) {
  /* computes the vector distance and euclidean norm of this vector distance
   * to a given star k.
   */
  
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  
  if (i < N_STARS) {
    DISTANCES[i].x = GALAXY[i].position.x - GALAXY[k].position.x;
    DISTANCES[i].y = GALAXY[i].position.y - GALAXY[k].position.y;
    DISTANCES[i].z = GALAXY[i].position.z - GALAXY[k].position.z;
    
    DISTANCES[i].l = LENGTH3D(DISTANCES[i].x, DISTANCES[i].y, DISTANCES[i].z);
  }
}

// ------------------------------------------------------------------------- //
void makeDistanceVector(unsigned int k) {
  if (k > N_stars) {
    fprintf(
      stderr,
      "%s: Invalid index %u\n",
      __func__, k
    );
    return;
  }
  
  makeDistanceComponent<<<nBlocks, blockSize>>>(k);
  hipDeviceSynchronize();
}

// ========================================================================= //
// get all distances from origin for all stars

__global__ void makeModulusVectorComponent(
  float *  dst,
  action_t action
) {
  /* computes the vector distance and euclidean norm of this vector distance
   * to a given star k.
   */
  
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  
  if (i < N_STARS) {
    switch(action) {
      case position_action :
        dst[i] = LENGTH3D(GALAXY[i].position.x, GALAXY[i].position.y, GALAXY[i].position.z);
        
      case velocity_action :
        dst[i] = LENGTH3D(GALAXY[i].velocity.x, GALAXY[i].velocity.y, GALAXY[i].velocity.z);
    }
  }
}

// ------------------------------------------------------------------------- //
void makeModulusVector (action_t action) {
  makeModulusVectorComponent<<<nBlocks, blockSize>>>(d_moduli, action);
  hipDeviceSynchronize();
}
