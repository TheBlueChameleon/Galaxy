#include "hip/hip_runtime.h"
/* starDistribution.cu
 * 
 * this module deals with snapshots of the galaxy: generation, retrieval of 
 * data from device and reduction into radial histograms.
 */


// ========================================================================= //
// dependencies

#include <ctime>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hiprand.h>

#include "globals.h"
#include "starDistribution.h"


// ========================================================================= //
// make galaxy

// ------------------------------------------------------------------------- //
// proc device

__global__ void makeGalaxyOnDevice_component() {
  /* assumes that all components of GALAXY have been initialized in the range
   * 0.0f..1.0f and rescales them according to the global constants
   */
  
  
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  
  if (i < N_STARS) {
    GALAXY[i].position.x -= 0.5;
    GALAXY[i].position.y -= 0.5;
    GALAXY[i].position.z -= 0.5;
    
    GALAXY[i].position.x *= D_UNIVERSE;
    GALAXY[i].position.y *= D_UNIVERSE;
    GALAXY[i].position.z *= D_UNIVERSE;
    
    GALAXY[i].velocity.x -= 0.5;
    GALAXY[i].velocity.y -= 0.5;
    GALAXY[i].velocity.z -= 0.5;
    
    GALAXY[i].velocity.x *= V_INIT_MAX;
    GALAXY[i].velocity.z *= V_INIT_MAX;
    GALAXY[i].velocity.z *= V_INIT_MAX;
    
    GALAXY[i].mass *= M_STAR_MAX;
  }
}

// ------------------------------------------------------------------------- //
// proc host

void makeGalaxyOnDevice() {
  /* This will initialize a galaxy with uniform distribution of
   * - position
   * - velocity
   * - mass
   * of the stars. This proc will place values 0.0f..1.0f into the given fields
   * and makes use of the below kernel to rescale/translate them into a 
   * reasonable range.
   * 
   * useful link:
   *   https://docs.nvidia.com/cuda/hiprand/host-api-overview.html
   */
  
  if (!flag_rand_initialized) {ABORT_WITH_MSG("RNG not initialized.");}
  
  // run the RNG
  hiprandGenerateUniform(
    d_RNG_mem, 
    (float *) d_galaxy,                             // galaxy is of type star *. star is a struct of only floats. This cast is justifiable.
    N_stars * (sizeof(*d_galaxy) / sizeof(float))
  );
  CudaCheckError();
  
  makeGalaxyOnDevice_component<<<nBlocks, blockSize>>>();
  hipDeviceSynchronize();
}

// ========================================================================= //
// get back galaxy from device

void fetchGalaxyFromDevice() {
  /* TODO: make this asynchronous
   * for this, you'll need to transform h_galaxy to a page locked array
   * use hipHostMalloc, cf. script p.33f.
   * 
   * This assumes that h_galaxy and d_galaxy have been properly initialized.
   * The "CTor" init() in globals.cu takes care of this.
   */
  
  hipMemcpy(
    h_galaxy, 
    d_galaxy, 
    N_stars * sizeof(*h_galaxy),
    hipMemcpyDeviceToHost
  );
  CudaCheckError();
}

// ========================================================================= //
// get all distances from a given star at index k

// ------------------------------------------------------------------------- //
// proc device
__global__ void makeDistanceComponent(unsigned int k) {
  /* computes the vector distance and euclidean norm of this vector distance
   * to a given star k.
   */
  
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  
  if (i < N_STARS) {
    DISTANCES[i].x = GALAXY[i].position.x - GALAXY[k].position.x;
    DISTANCES[i].y = GALAXY[i].position.y - GALAXY[k].position.y;
    DISTANCES[i].z = GALAXY[i].position.z - GALAXY[k].position.z;
    
    DISTANCES[i].l = LENGTH3D(DISTANCES[i].x, DISTANCES[i].y, DISTANCES[i].z);
  }
}

// ------------------------------------------------------------------------- //
// proc host
void makeDistanceVector(unsigned int k) {
  if (k > N_stars) {
    fprintf(
      stderr,
      "%s: Invalid index %u\n",
      __func__, k
    );
    return;
  }
  
  makeDistanceComponent<<<nBlocks, blockSize>>>(k);
  hipDeviceSynchronize();
}

// ========================================================================= //
// get all distances from origin for all stars

// ------------------------------------------------------------------------- //
// proc device
__global__ void makeModulusVectorComponent(
  action_t action,
  float *  dst
) {
  /* computes the modulus for each star's position or velocity
   */
  
  unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
  
  if (i < N_STARS) {
    switch(action) {
      case position_action :
        dst[i] = LENGTH3D(GALAXY[i].position.x, GALAXY[i].position.y, GALAXY[i].position.z);
        
      case velocity_action :
        dst[i] = LENGTH3D(GALAXY[i].velocity.x, GALAXY[i].velocity.y, GALAXY[i].velocity.z);
    }
  }
}

// ------------------------------------------------------------------------- //
// proc host
void makeModulusVector (action_t action) {
  makeModulusVectorComponent<<<nBlocks, blockSize>>>(action, d_moduli);
  hipDeviceSynchronize();
  
  hipMemcpy(
    h_moduli,
    d_moduli, 
    N_stars * sizeof(*h_moduli),
    hipMemcpyDeviceToHost
  );
  CudaCheckError();
}

// ========================================================================= //
// recenter galaxy in centre of mass and make average velocity = (0,0,0)

// ------------------------------------------------------------------------- //
// proc device

__global__ void copyWeightedComponent(
  action_t     action,
  vector3D_t * dst
) {
  unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
  
  if (i < N_STARS) {
    switch(action) {
      case position_action :
        dst[i].x = GALAXY[i].position.x * GALAXY[i].mass / N_STARS;
        dst[i].y = GALAXY[i].position.y * GALAXY[i].mass / N_STARS;
        dst[i].z = GALAXY[i].position.z * GALAXY[i].mass / N_STARS;
        
      case velocity_action :
        dst[i].x = GALAXY[i].velocity.x * GALAXY[i].mass / N_STARS;
        dst[i].y = GALAXY[i].velocity.y * GALAXY[i].mass / N_STARS;
        dst[i].z = GALAXY[i].velocity.z * GALAXY[i].mass / N_STARS;
    }
  }
}
// ......................................................................... //
__global__ void reduction_galaxyCentre(
  action_t     action,
  vector3D_t * dst
  
) {
  
}
// ......................................................................... //
__global__ void translateComponent(
  action_t   action,
  vector3D_t offset
) {
  unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
  
  if (i < N_STARS) {
    switch(action) {
      case position_action :
        GALAXY[i].position.x += offset.x;
        GALAXY[i].position.y += offset.y;
        GALAXY[i].position.z += offset.z;
        
      case velocity_action :
        GALAXY[i].velocity.x += offset.x;
        GALAXY[i].velocity.y += offset.y;
        GALAXY[i].velocity.z += offset.z;
    }
  }
}

// ------------------------------------------------------------------------- //
// proc host
void makeCentered() {
  /* This drives a reduction as for re-centering, the centre is needed, obviously.
   * Reduction is run on a copy of the galaxy's position and velocity coordinates.
   * 
   * In the following comments, COM stands for centre of mass, while COP 
   * represents centre of momentum.
   */
  
  vector3D_t  centreOfMass, 
              centreOfMomentum,
              * d_positions  = nullptr,     // temp device arrays 
              * d_velocities = nullptr;     // do reductions on these
  
  
  // get memory for reduction to COM & COP
  hipMalloc(&d_positions , N_stars * sizeof(*d_positions));
  if (!d_positions ) {ABORT_WITH_MSG("position reduction device memory not initialized.");}
  CudaCheckError();
  
  hipMalloc(&d_velocities, N_stars * sizeof(*d_velocities));
  if (!d_velocities) {ABORT_WITH_MSG("velocity reduction device memory not initialized.");}
  CudaCheckError();
  
  
  // copy to buffer COM & COP with weight mass[i]/N_stars
  copyWeightedComponent<<<nBlocks, blockSize>>>(position_action, d_positions );
  copyWeightedComponent<<<nBlocks, blockSize>>>(velocity_action, d_velocities);
  
  
  // get COM & COP
  // sync
  
  
  // free buffer COM & COP
  if (d_positions ) {hipFree(d_positions ); CudaCheckError();}
  if (d_velocities) {hipFree(d_velocities); CudaCheckError();}
  
  
  // translate by COM & COP
  translateComponent<<<nBlocks, blockSize>>>(position_action, centreOfMass    );
  translateComponent<<<nBlocks, blockSize>>>(velocity_action, centreOfMomentum);
}
